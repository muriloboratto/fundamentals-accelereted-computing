/*
@(#)File:           $mini-nbody-parallel.cu$
@(#)Version:        $Parallel version N-Body Simulation$
@(#)Last changed:   $Date: 20220319 11:13:00 $
@(#)Purpose:        Fast N-Body Simulation
@(#)Author:         Murilo Boratto - muriloboratto@gmail.com
@(#)Usage:         
 (*) Hotocompile:   nvcc mini-nbody-parallel.cu -o mini-nbody-parallel -Xcompiler -fopenmp
 (*) Hotoexecute:   ./mini-nbody-parallel <size nBodies>
@(#)Comment:  
    (*) [1] https://developer.nvidia.com/gpugems/gpugems3/part-v-physics-simulation/chapter-31-fast-n-body-simulation-cuda            
    (*) [2] https://github.com/harrism/mini-nbody/blob/master/nbody.c
*/


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__
void bodyForce(Body *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

int main(const int argc, const char** argv) {
  
  int nBodies = 30000;
  
  if (argc > 1) 
    nBodies = atoi(argv[1]);
  
  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations
  int bytes = nBodies*sizeof(Body);
  float *buf = (float*)malloc(bytes);
  Body *p = (Body*)buf;

  randomizeBodies(buf, 6*nBodies); // Init pos / vel data

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  Body *d_p = (Body*)d_buf;
  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 

  const double t3 = omp_get_wtime();

  for (int iter = 1; iter <= nIters; iter++) {
    const double t1 = omp_get_wtime();

    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
          bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p, dt, nBodies); // compute interbody forces
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double t2 = omp_get_wtime();
    totalTime += (t2-t1); 
    //printf("Iteration %d: %.3f seconds\n", iter, totalTime);
  }
  const double t4 = omp_get_wtime();

  double avgTime = (t4-t3) / (double)(nIters-1); 
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
  printf("\n Size (Bodies) = %d\n", nBodies);
  printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);
  printf("%0.3f second\n", avgTime);
  
  free(buf);
  hipFree(d_buf);

  return 0;
}