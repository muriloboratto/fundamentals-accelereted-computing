#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;

__global__
void bodyForce(Body *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}


int main(const int argc, const char** argv) {

  int nBodies = 2<<11;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  const char * initialized_values;
  const char * solution_values;

  if (nBodies == 2<<11) {
    initialized_values = "09-nbody/files/initialized_4096";
    solution_values = "09-nbody/files/solution_4096";
  } else { // nBodies == 2<<15
    initialized_values = "09-nbody/files/initialized_65536";
    solution_values = "09-nbody/files/solution_65536";
  }

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  const float dt = 0.01f; // Time step
  const int nIters = 10;  // Simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;

  buf = (float *)malloc(bytes);
  Body *p = (Body*)buf;

  read_values_from_file(initialized_values, buf, bytes);

  double totalTime = 0.0;

// CHANGE ===========================

  int size_in_bytes = sizeof(Body) * nBodies;
 
  float *device_buf;
  hipMalloc((void**) &device_buf, size_in_bytes);
  
  Body *d_p = (Body*)device_buf;
    
  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  int multiProcessorCount = props.multiProcessorCount;
   
  size_t threadsPerBlock = 256;
  size_t numberOfBlocks = 32 * multiProcessorCount;

// CHANGE ===========================


  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();

    hipMemcpy(device_buf, buf, size_in_bytes, hipMemcpyHostToDevice);
       bodyForce<<< numberOfBlocks, threadsPerBlock>>>(d_p, dt, nBodies); // compute interbody forces
    hipMemcpy(buf, device_buf, size_in_bytes, hipMemcpyDeviceToHost);
  
    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
  write_values_to_file(solution_values, buf, bytes);

  printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

  free(buf);
  hipFree(device_buf);
}
